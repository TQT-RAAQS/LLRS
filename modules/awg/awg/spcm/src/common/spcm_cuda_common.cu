// #include "spcm_cuda_common.h"
#include "common/spcm_cuda_common.h"

#include <cstdio>

// ----- CUDA include -----
#include <hip/hip_runtime.h>

// CUDA-C includes
#include <hip/hip_runtime.h>

// ----- Init CUDA device without RDMA support (e.g. on Windows). -----
// -- lCUDADevIdx:  index of CUDA device to be used
// -- return:       true if initialization succeeded, false otherwise
bool bInitCUDADevice(int lCUDADevIdx) {
    // ----- check for CUDA-capable devices -----
    int lCUDADeviceCount = 0;
    hipError_t eCudaErr = hipGetDeviceCount(&lCUDADeviceCount);
    if (eCudaErr != hipSuccess) {
        printf("ERROR in hipGetDeviceCount(): %s\n",
               hipGetErrorString(eCudaErr));
        return false;
    }

    if (lCUDADeviceCount == 0) {
        printf("ERROR: there are no available devices that support CUDA\n");
        return false;
    }
    if (lCUDADevIdx >= lCUDADeviceCount) {
        printf("ERROR: requested device %d, but only %d CUDA device(s) "
               "available\n",
               lCUDADevIdx, lCUDADeviceCount);
        return false;
    }

    printf("Detected %d CUDA Capable device(s).\n", lCUDADeviceCount);
    hipSetDevice(lCUDADevIdx);
    hipDeviceProp_t stCUDADeviceProp;
    hipGetDeviceProperties(&stCUDADeviceProp, lCUDADevIdx);

    printf("\nUsing device %d: \"%s\"\n", 0, stCUDADeviceProp.name);

    return true;
}

#ifndef WIN32

// ----- Sets the CUDA device (GPU) to be used and allocates a buffer that's
// usable for RDMA. -----
// -- lCUDADevIdx:           index of CUDA device to be used
// -- qwDMABufferSize_bytes: size of the buffer that should be allocated
// -- return:                pointer to buffer of requested size if
// initialization succeeded, NULL otherwise
void *pvGetRDMABuffer(int lCUDADevIdx, size_t qwDMABufferSize_bytes) {
    hipError_t eCudaErr;
    //     // ----- check for CUDA-capable devices -----
    //     int lCUDADeviceCount = 0;
    //     eCudaErr = hipGetDeviceCount (&lCUDADeviceCount);
    //     if (eCudaErr != hipSuccess)
    //         {
    //         printf ("ERROR in hipGetDeviceCount(): %s\n",
    //         hipGetErrorString(eCudaErr)); return NULL;
    //         }

    //     if (lCUDADeviceCount == 0)
    //         {
    //         printf ("ERROR: there are no available devices that support
    //         CUDA\n"); return NULL;
    //         }
    //     if (lCUDADevIdx >= lCUDADeviceCount)
    //         {
    //         printf ("ERROR: requested device %d, but only %d CUDA device(s)
    //         available\n", lCUDADevIdx, lCUDADeviceCount); return NULL;
    //         }

    //     printf ("Detected %d CUDA Capable device(s).\n", lCUDADeviceCount);
    //     hipSetDevice (lCUDADevIdx);
    //     hipDeviceProp_t stCUDADeviceProp;
    //     hipGetDeviceProperties (&stCUDADeviceProp, lCUDADevIdx);
    //     if ((strncmp (stCUDADeviceProp.name, "Quadro", 6) != 0) && (strncmp
    //     (stCUDADeviceProp.name, "Tesla", 5) != 0))
    //         {
    //         printf ("ERROR: found \"%s\", but RDMA requires a Quadro or Tesla
    //         card.\n", stCUDADeviceProp.name); return NULL;
    //         }
    //     printf("\nUsing device %d: \"%s\"\n", 0, stCUDADeviceProp.name);

    //     // ----- we require at least CUDA 5.0 -----
    //     if (stCUDADeviceProp.major < 5)
    //         {
    //         printf ("ERROR: RDMA requires at least CUDA compute
    //         capability 5.0 (found: %d.%d)\n", stCUDADeviceProp.major,
    //         stCUDADeviceProp.minor); return NULL;
    //         }

    // ----- allocate DMA buffer on GPU -----
    // bInitCUDADevice (0);
    void *pvDMABuffer_gpu;
    eCudaErr = hipMalloc((void **)&pvDMABuffer_gpu, qwDMABufferSize_bytes);
    if (eCudaErr != hipSuccess) {
        printf("ERROR in hipMalloc(): %s\n", hipGetErrorString(eCudaErr));
        return NULL;
    }

    /*
    eCudaErr = hipMalloc((hipDeviceptr_t *)&pvDMABuffer_gpu,
    qwDMABufferSize_bytes); if (eCudaErr != hipSuccess)
        {
        printf ("ERROR in hipMalloc(): %s\n", hipGetErrorString(eCudaErr));
        return NULL;
        }*/

    // in GPUDirect RDMA scope should always be HIP_POINTER_ATTRIBUTE_SYNC_MEMOPS
    unsigned int dwFlag = 1;
    hipError_t eResult = hipPointerSetAttribute(
        (void *)&dwFlag,
        (hipPointer_attribute)HIP_POINTER_ATTRIBUTE_SYNC_MEMOPS,
        (hipDeviceptr_t)pvDMABuffer_gpu);
    if (eResult != hipSuccess) {
        printf("eResult: %d\n", (int)eResult);
        const char *szError;
        hipDrvGetErrorString(eResult, &szError);
        printf("ERROR in hipPointerSetAttribute(ATTRIBUTE_SYNC_MEMOPS): %s\n",
               szError);
        hipFree((void *)pvDMABuffer_gpu);
        return NULL;
    }

    return (void *)pvDMABuffer_gpu;
}

#endif

// ----- Returns error description for CUDA FFT error code -----
const char *szCudaGetErrorText(hipfftResult eError) {
    switch (eError) {
    case HIPFFT_SUCCESS:
        return "HIPFFT_SUCCESS";
    case HIPFFT_INVALID_PLAN:
        return "HIPFFT_INVALID_PLAN";
    case HIPFFT_ALLOC_FAILED:
        return "HIPFFT_ALLOC_FAILED";
    case HIPFFT_INVALID_TYPE:
        return "HIPFFT_INVALID_TYPE";
    case HIPFFT_INVALID_VALUE:
        return "HIPFFT_INVALID_VALUE";
    case HIPFFT_INTERNAL_ERROR:
        return "HIPFFT_INTERNAL_ERROR";
    case HIPFFT_EXEC_FAILED:
        return "HIPFFT_EXEC_FAILED";
    case HIPFFT_SETUP_FAILED:
        return "HIPFFT_SETUP_FAILED";
    case HIPFFT_INVALID_SIZE:
        return "HIPFFT_INVALID_SIZE";
    case HIPFFT_UNALIGNED_DATA:
        return "HIPFFT_UNALIGNED_DATA";
    }

    return "<unknown>";
}
