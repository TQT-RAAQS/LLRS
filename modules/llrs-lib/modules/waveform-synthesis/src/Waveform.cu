#include "hip/hip_runtime.h"
#include <Waveform.h>

std::vector<double> Synthesis::Waveform::discretize(size_t sample_rate) {

    std::vector<double> data;
    double time;
    for (int i = 0; i < duration * sample_rate; i++) {
        time = i / sample_rate;
        data.push_back(wave_func(time));
    }
    return data;
}

double Synthesis::Extraction::wave_func(double time) {
    std::make_tuple<double, double, double> srcTuples(0, std::get<1>(params),
                                                      std::get<2>(params));

    return Synthesis::Waveform::transMod->transition_func(t, srcTuples, params);
}

double Synthesis::Implantation::wave_func(double time) {
    std::make_tuple<double, double, double> destParams(0, std::get<1>(params),
                                                       std::get<2>(params));

    return Synthesis::Waveform::transMod->transition_func(t, params,
                                                          destParams);
}

double Synthesis::Idle::wave_func(std::size_t index) {
    return Synthesis::Waveform::staticMod->static_func(t, params);
}

double Synthesis::Displacement::wave_func(std::size_t index) {
    return Synthesis::Waveform::transMod->transition_func(t, params,
                                                          destParams);
}

// ------------------ WAVEFORM FUNCTIONS ----------------

double Sin::transition_func(double t, WP params) {
    double alpha, nu, phi;
    std::tie(alpha, nu, phi) = params;

    return alpha * sin(2 * M_PI * nu * t + phi);
}

double ERF::transition_func(double t, WP params1, WP params2) {
    double alpha1, nu1, phi1;
    double alpha2, nu2, phi2;
    std::tie(alpha0, nu0, phi0) = params1;
    std::tie(alpha1, nu1, phi1) = params2;

    double alpha_mean = (alpha1 + alpha2) / 2;
    double dalpha = alpha2 - alpha_1;
    double alpha = alpha_mean + dalpha / 2 *
                                    erf(sqrt(M_PI) * vmax * (t - duration / 2);)

                                        double nu_mean = (nu1 + nu2) / 2;
    double dnu = nu2 - nu1;
    double phi_tilde =
        nu_mean * t +
        dnu / (2 * sqrt(M_PI) * vmax) *
            (sqrt(M_PI) * vmax * (t - duration / 2) *
                 erf(sqrt(M_PI) * vmax * (t - duration / 2)) +
             exp(-pow(sqrt(M_PI) * vmax * (t - duration / 2), 2)) / sqrt(M_PI) -
             sqrt(M_PI) * vmax * duration / 2 *
                 erf(sqrt(M_PI) * vmax * duration / 2) -
             exp(-pow(sqrt(M_PI) * vmax * (duration / 2), 2)) / sqrt(M_PI))

                double dphi = fmod(phi2 - phi1, 2 * M_PI);
    dphi = dphi - (abs(dphi) > M_PI) * (2 * (dphi > 0) - 1) * 2 * M_PI;

    return alpha * sin(phi1 + 2 * M_PI * phi_tilde + dphi * t / duration);
}

double TANH::transition_func(double t, WP params1, WP params2) {
    double alpha1, nu1, phi1;
    double alpha2, nu2, phi2;
    std::tie(alpha0, nu0, phi0) = params1;
    std::tie(alpha1, nu1, phi1) = params2;

    double alpha_mean = (alpha1 + alpha2) / 2;
    double dalpha = alpha2 - alpha_1;
    double alpha = alpha_mean + dalpha / 2 *
                                    tanh(2 * vmax * (t - duration / 2);)

                                        double nu_mean = (nu1 + nu2) / 2;
    double dnu = nu2 - nu1;
    double phi_tilde =
        nu_mean * t +
        dnu / (4 * vmax) *
            (log(cosh(2 * vmax * (t - T / 2))) - log(cosh(vmax * T)))

                double dphi = fmod(phi2 - phi1, 2 * M_PI);
    dphi = dphi - (abs(dphi) > M_PI) * (2 * (dphi > 0) - 1) * 2 * M_PI;

    return alpha * sin(phi1 + 2 * M_PI * phi_tilde + dphi * t / duration);
}

double Spline::transition_func(double t, WP params1, WP params2) {
    double alpha1, nu1, phi1;
    double alpha2, nu2, phi2;
    std::tie(alpha0, nu0, phi0) = params1;
    std::tie(alpha1, nu1, phi1) = params2;

    double a1 = -2 * (alpha2 - alpha1) * pow(t / duration, 3);
    double b1 = 3 * (alpha2 - alpha1) * pow(t / duration, 2);
    double alpha = a1 + b1 + alpha1;

    double a2 = -(nu2 - nu1) / (2 * pow(T, 3)) * pow(t, 4);
    double b2 = (nu2 - nu1) / (pow(T, 2)) * pow(t, 3);
    double phi_tilde = a2 + b2 + nu1 * t;

    double dphi = fmod(phi2 - phi1 - phi_tilde, 2 * M_PI);
    dphi = dphi - (abs(dphi) > M_PI) * (2 * (dphi > 0) - 1) * 2 * M_PI;

    return alpha * sin(phi1 + 2 * M_PI * phi_tilde + dphi * t / duration);
}

void read_waveform_configs(std::string filepath) {
    /// Open file
    try {
        YAML::Node node = YAML::LoadFile(filename);
    } catch (const YAML::BadFile &e) {
        std::cerr << "Error loading YAML file (Waveform Config)." << std::endl;
        std::cerr << "ERROR: " << e.what() << std::endl;
        return;
    }

	double waveform_duration = node["waveform_duration"].as<double>();

    // Transition Mod
    std::string transition_type = node["transition_type"].as<std::string>();
    if (transition_type == "TANH") {
        Waveform::set_transition_function(
            std::make_unique<TANH>(waveform_duration, node["transition_type"]["vmax"].as<double>()));
    } else if (transition_type == "Spline") {
        Waveform::set_transition_function(
            std::make_unique<Spline>(waveform_duration));
    } else if (transition_type == "Step") {
        Waveform::set_transition_function(
            std::make_unique<Step>(waveform_duration));
    } else if (transition_type == "ERF") {
        Waveform::set_transition_function(
            std::make_unique<ERF>(waveform_duration, node["transition_type"]["vmax"].as<double>()));   } else {
        throw std::invalid_argument(
            "Transition modulation type not supported.");
    }

    // Static Mod
    std::string static_type == node["static_type"].as<std::string>();
    Waveform::staticMod = std::make_unique<Spline>(node["static_type"]);
    if (transition_type == "sin") {
        Waveform::set_static_function(
            std::make_unique<Sin>());
    } else {
        throw std::invalid_argument("Static modulation type not supported.");
    }
}

double read_waveform_duration(std::string filepath) {
    /// Open file
    try {
        YAML::Node node = YAML::LoadFile(filename);
    } catch (const YAML::BadFile &e) {
        std::cerr << "Error loading YAML file (Waveform Config)." << std::endl;
        std::cerr << "ERROR: " << e.what() << std::endl;
        return;
    }

	return node["waveform_duration"].as<double>();
}
