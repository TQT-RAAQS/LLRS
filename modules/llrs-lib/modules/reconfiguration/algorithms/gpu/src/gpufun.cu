#include "kernel.h"
#include <chrono>

double solve_gpu(int *sourceFlags, int *targetFlags, int numTraps, int numSources,
          int numTargets, int *OutSources_cpu, int *OutTargets_cpu) {
    // Allocate memory
    int *sourceFlags_d, *targetFlags_d, *OutSources_gpu_d, *OutTargets_gpu_d;
    int numExcessSources = numSources - numTargets;
    hipMalloc((void **)&sourceFlags_d, numTraps * sizeof(int));
    hipMalloc((void **)&targetFlags_d, numTraps * sizeof(int));
    hipMalloc((void **)&OutSources_gpu_d, numTargets * sizeof(int));
    hipMalloc((void **)&OutTargets_gpu_d, numTargets * sizeof(int));
    hipDeviceSynchronize();
    // Copy data to GPU
    hipMemcpy(sourceFlags_d, sourceFlags, numTraps * sizeof(int),
               hipMemcpyHostToDevice);
    hipMemcpy(targetFlags_d, targetFlags, numTraps * sizeof(int),
               hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    
    // Compute on GPU
    auto t1 = std::chrono::high_resolution_clock::now();
    solve_gpu_d(numExcessSources, sourceFlags_d, targetFlags_d, numTraps,
                OutSources_gpu_d, OutTargets_gpu_d);
    hipDeviceSynchronize();
    auto t2 = std::chrono::high_resolution_clock::now();
    double computeTime = std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count();
    // Copy data from GPU
    hipMemcpy(OutSources_cpu, OutSources_gpu_d, numTargets * sizeof(int),
               hipMemcpyDeviceToHost);
    hipMemcpy(OutTargets_cpu, OutTargets_gpu_d, numTargets * sizeof(int),
               hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    
    // Free memory
    hipFree(sourceFlags_d);
    hipFree(targetFlags_d);
    hipFree(OutSources_gpu_d);
    hipFree(OutTargets_gpu_d);
    hipDeviceSynchronize();
    return computeTime;
}