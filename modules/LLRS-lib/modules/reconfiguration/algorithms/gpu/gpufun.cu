#include <ctime>
#include <chrono>
#include "gpufun.h"
#include "kernel.h"
#include "timer.h"

//#define VERBOSE 

//#define PRINT_GPU_RUNTIME


std::chrono::duration<double, std::milli> solve_gpu(int* sourceFlags, int* targetFlags, int numTraps, int numSources, int numTargets, int* OutSources_gpu, int* OutTargets_gpu)
{
    Timer timer;
	//float computeTimer;

    std::chrono::time_point<std::chrono::high_resolution_clock> t1;
    std::chrono::time_point<std::chrono::high_resolution_clock> t2;

    std::chrono::duration<double, std::milli> computeTimer;
	
    // Allocate memory
    startTime(&timer);
    int *sourceFlags_d, *targetFlags_d, *OutSources_gpu_d, *OutTargets_gpu_d;
    int numExcessSources = numSources - numTargets;
    hipMalloc((void**) &sourceFlags_d, numTraps*sizeof(int));
    hipMalloc((void**) &targetFlags_d, numTraps*sizeof(int));
    hipMalloc((void**) &OutSources_gpu_d, numTargets*sizeof(int));
    hipMalloc((void**) &OutTargets_gpu_d, numTargets*sizeof(int));
    hipDeviceSynchronize();
    stopTime(&timer);
#ifdef VERBOSE
    printElapsedTime(timer, "\nMy Allocation time", DGREEN);
#endif
    // Copy data to GPU
    startTime(&timer);
    hipMemcpy(sourceFlags_d, sourceFlags, numTraps*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(targetFlags_d, targetFlags, numTraps*sizeof(int), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    stopTime(&timer);

#ifdef VERBOSE
    printElapsedTime(timer, "My Copy to GPU time", DGREEN);
#endif


    // Compute on GPU
    t1 = std::chrono::high_resolution_clock::now();
    solve_gpu_d(numExcessSources, sourceFlags_d, targetFlags_d, numTraps, OutSources_gpu_d, OutTargets_gpu_d);
    hipDeviceSynchronize();
	t2 = std::chrono::high_resolution_clock::now();
    computeTimer = t2 - t1;
#ifdef PRINT_GPU_RUNTIME
    printElapsedTime(timer, "GPU time is:", CYAN);
#endif

    // Copy data from GPU
    startTime(&timer);
    hipMemcpy(OutSources_gpu, OutSources_gpu_d, numTargets*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(OutTargets_gpu, OutTargets_gpu_d, numTargets*sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    stopTime(&timer);
#ifdef VERBOSE
    printElapsedTime(timer, "My Copy from GPU time", DGREEN);
#endif
    // Free memory
    startTime(&timer);
    hipFree(sourceFlags_d);
    hipFree(targetFlags_d);
    hipFree(OutSources_gpu_d);
	hipFree(OutTargets_gpu_d);
    hipDeviceSynchronize();
    stopTime(&timer);
#ifdef VERBOSE
    printElapsedTime(timer, "Deallocation time", DGREEN);
#endif
	
	return computeTimer;

}