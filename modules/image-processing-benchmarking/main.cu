#include "hip/hip_runtime.h"
#include <ctime>
#include <chrono>
#include <cmath>
#include <numeric>
#include "ImageProcessor.h"

int reps = 100;
int N_t = 32;
int kernel_size = 1;
std::string psf_path {};

// function to take in command line args for height and psf_path, width stays constant
int cmd_line(int argc, char * argv[]) 
{
    if (argc != 5) {
        std::cout<< "Usage is: ./image_processing_unit <psf_path> <Nt> <kernel_size> <reps>" << std::endl;
        return 1;
    }
    psf_path = std::string(argv[1]);
    N_t = std::stoi(argv[2]);
    kernel_size = std::stoi(argv[3]);
    reps = std::stoi(argv[4]);
    return 0;
}

int main(int argc, char * argv[]) {

    // taking in command line arguments
    if (cmd_line(argc, argv) != 0) {
        return 1;
    }

    // creating vector to store all times
    std::vector<double> results;
	results.reserve(reps);

    // initialize Image Processing object 
    Processing::ImageProcessor img_proc_obj(
       psf_path, N_t
    );

    // 
    for (int i = 0; i < reps; ++i) {
        // initialize vector for current image
        std::vector<uint16_t> current_image(N_t * kernel_size * kernel_size, 0);

        // arbitrarily change the values for the vector of pixels
        for (int j = 0; j < N_t * kernel_size * kernel_size; ++j){
            current_image[j] = j % 16;
        }

        std::chrono::steady_clock::time_point begin;
        std::chrono::steady_clock::time_point end;

        begin = std::chrono::steady_clock::now();
        std::vector<double> filtered_output = img_proc_obj.apply_filter(&current_image);
        end = std::chrono::steady_clock::now();

        std::chrono::duration<double> time_span = std::chrono::duration_cast<std::chrono::duration<double>>(end - begin);
        results.push_back(time_span.count());
    }

	double average = std::accumulate(results.begin(), results.end(), 0.0) / reps;
	std::vector<double> diffs;
	diffs.reserve(results.size());
	for (auto it: results) {
		diffs.push_back(it - average);
	}
	double stddev  = std::sqrt(std::inner_product(diffs.begin(), diffs.end(), diffs.begin(), 0.0) / (reps - 1));
    std::cout << average << std::endl << stddev << std::endl;

}
